#include <stdio.h>
#include <hip/hip_runtime.h>

typedef float MatrixVal;

typedef struct matrix {
	MatrixVal *values;
	unsigned int rows, cols;
} Matrix;

typedef struct input {
	Matrix *A, *B;
} Input;

void setMatrixPosition(Matrix *matrix, unsigned int row, unsigned int col, MatrixVal value) {
	matrix->values[col + matrix->cols * row] = value;
}

MatrixVal getMatrixPosition(Matrix *matrix, unsigned int row, unsigned int col) {
	return matrix->values[col + matrix->cols * row];
}

void setMatrixSize(Matrix *matrix, unsigned int rows, unsigned int cols) {
	matrix->values = (MatrixVal *) malloc(rows * cols * sizeof(MatrixVal));
	matrix->cols = cols;
	matrix->rows = rows;
}

Matrix *newMatrix() {
	Matrix *matrix = (Matrix *) malloc(sizeof(Matrix));
	return matrix;
}

void deleteMatrix(Matrix *matrix) {
	free(matrix->values);
	free(matrix);
}

Matrix *readMatrixFrom(FILE *src) {
	unsigned int row, col, rows, cols;
	MatrixVal value;
	Matrix *matrix = newMatrix();
	fscanf(src, "%u %u", &rows, &cols);
	setMatrixSize(matrix, rows, cols);
	for (row = 0; row < rows; row++) {
		for (col = 0; col < cols; col++) {
			fscanf(src, "%f", &value);
			setMatrixPosition(matrix, row, col, value);
		}
	}
	return matrix;
}

void deleteInput(Input input) {
	deleteMatrix(input.A);
	deleteMatrix(input.B);
}

Input readMatricesFromFiles(char *fileName1, char *fileName2) {
	Input input;
	FILE *file1, *file2;
	file1 = fopen(fileName1, "r");
	input.A = readMatrixFrom(file1);
	fclose(file1);
	file2 = fopen(fileName2, "r");
	input.B = readMatrixFrom(file2);
	fclose(file2);
	return input;
}

Input readMatricesFromStdin() {
	Input input;
	input.A = readMatrixFrom(stdin);
	input.B = readMatrixFrom(stdin);
	return input;
}

void printUsage() {
	printf("Usage: matrix-multiply <cuda|cpu> [file-with-matrix1 file-with-matrix2]\n");
	printf("\nIf files are not passed, matrices are read from stdin.\n");
	printf("Input format: n-rows n-cols entries\n");
	printf("Output format: n-rows n-cols result-entries\n");
	printf("Output is always to stdout\n");
}

void processUsingCuda(Input input) {
}

void processUsingCpu(Input input) {
}

int main(int argc, char **argv) {
	Input input;
	if (argc == 2) {
		input = readMatricesFromStdin();
	} else if (argc == 4) {
		input = readMatricesFromFiles(argv[2], argv[3]);
	} else {
		printf("Error: wrong number of arguments: %d\n", argc);
		printUsage();
		return 1;
	}
	if (strcmp(argv[1], "cuda") == 0) {
		processUsingCuda(input);
	} else if (strcmp(argv[1], "cpu") == 0) {
		processUsingCpu(input);
	} else {
		printf("Error: %s is not a valid form of computation\n");
		printUsage();
		return 2;
	}
	return 0;
}
